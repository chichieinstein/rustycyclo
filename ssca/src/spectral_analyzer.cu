#include "hip/hip_runtime.h"
#include "../include/spectral_analyzer.cuh"
#include <hipfft/hipfft.h>
#include <iostream>
#include <chrono>
#include <cmath>
#include <complex.h>
#include <math.h>
#include <iomanip>
#include <stdio.h>

using std::cout;
using std::endl;
const float pi = acos(-1);
void __global__ make_conj(hipfftComplex *in, size_t size)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
        in[id] = make_hipComplex(in[id].x, -in[id].y);
}

// void __global__ fft_shift(bool stride, hipfftComplex *inp, int N, int Np)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int id = idy * Np + idx;
//     int sign;
//     if ((idy < N) && (idx < Np))
//     {
//         if (stride)
//         {
//             sign = 1 - 2 * (id % 2);
//         }
//         else
//         {
//             int row_pos = (int)id / Np;
//             sign = 1 - 2 * (row_pos % 2);
//         }
//         inp[id] = make_hipComplex(inp[id].x * static_cast<float>(sign), inp[id].y * static_cast<float>(sign));
//     }
// }

// void __global__ vect_multiply(hipfftComplex *in, hipfftComplex *vec, bool row_wise, bool conj, int N, int Np)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.x;
//     int id = idy * Np + idx;
//     int vec_ind;
//     if ((idx < Np) && (idy < N))
//     {
//         if (row_wise)
//             vec_ind = idx;
//         else
//             vec_ind = idy;

//         if (conj)
//             in[id] = make_hipComplex(in[id].x * vec[vec_ind].x + in[id].y * vec[vec_ind].y, -in[id].x * vec[vec_ind].y + in[id].y * vec[vec_ind].x);
//         else
//         {
//             // printf("Real part of input : %f, Imaginary part of input : %f, real part of vec : %f, imag part of vec : %f , prod_real : %f, prod_imag : %f\n", in[id].x, in[id].y, vec[vec_ind].x, vec[vec_ind].y, in[id].x * vec[vec_ind].x - in[id].y * vec[vec_ind].y, in[id].x * vec[vec_ind].y + in[id].y * vec[vec_ind].x );
//             in[id] = make_hipComplex(in[id].x * vec[vec_ind].x - in[id].y * vec[vec_ind].y, in[id].x * vec[vec_ind].y + in[id].y * vec[vec_ind].x);
//         }
//     }
// }

// void __global__ mat_multiply(hipfftComplex *left, hipfftComplex *right, int N, int Np)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int id = idy * Np + idx;
//     // int id = blockIdx.x * blockDim.x + threadIdx.x;
//     if ((idy < N) && (idx < Np))
//         right[id] = make_hipComplex(left[id].x * right[id].x - left[id].y * right[id].y, left[id].x * right[id].y + left[id].y * right[id].x);
// }

// void __global__ average(hipfftComplex *in, float *out, int N, int Np, int BATCH)
// {
//     int id = blockIdx.x * blockDim.x + threadIdx.x;
//     if (id < N * Np)
//     {
//         for (int idx = 0; idx < BATCH; idx++)
//             out[id] += hypotf(in[idx * N * Np + id].x, in[idx * N * Np + id].y);
//         out[id] = out[id] / static_cast<float>(BATCH);
//     }
// }

void __global__ batch_average(hipfftComplex *in, float *out, int N, int Np, int BATCH)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    int id2d = idy * Np + idx;

    if ((idx < Np) && (idy < N) && (idz < BATCH))
    {
        out[id2d] += hypotf(in[idz * N * Np + id2d].x, in[idz * N * Np + id2d].y);
    }
}

void __global__ set_zero(float *in, int max_size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < max_size)
    {
        in[id] = static_cast<float>(0);
    }
}

void __global__ reductor(float *in, float *out, int N, int Np, int max_size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // int max_size    = 2 * N - (Np / 2);
    if (id < max_size)
    {
        int reduced_index = (id < N) ? id : id - N + (Np / 2);
        // cycles[id]        = (id < N) ? Q[reduced_index] + K[0] : Q[reduced_index] + K[Np-1];
        int quotient = (2 * reduced_index) / Np;
        if (id < N)
        {
            for (int new_ind = 0; new_ind <= quotient; new_ind++)
            {
                int xind = id - (new_ind * Np) / 2;
                out[id] += in[xind * Np + new_ind];
            }
        }
        else
        {
            int counter = (Np - 1) - quotient;
            for (int new_ind = 0; new_ind < counter; new_ind++)
            {
                int xind = reduced_index + (new_ind * Np) / 2;
                out[id] += in[xind * Np + Np - 1 - new_ind];
            }
        }
    }
}

// void __global__ create_matrix_gpu(hipfftComplex *in, hipfftComplex *out, int N, int Np)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;

//     if ((idx < Np) && (idy < N))
//     {
//         out[idy * Np + idx] = in[idx + idy];
//     }
// }

// void create_batch_matrix_gpu(hipfftComplex *input, hipfftComplex *output, int N, int Np, int BATCH)
// {
//     int jump{(N + Np) / 2};
//     int ind;
//     dim3 dimBlock(32, 32);
//     dim3 dimGrid(N / 32, Np / 32);
//     for (ind = 0; ind < BATCH; ind++)
//         create_matrix_gpu<<<dimGrid, dimBlock>>>(input + ind * jump, output + ind * N * Np, N, Np);
// }

// void __global__ create_batch_matrix_gpu_3d(hipfftComplex *input, hipfftComplex *output, int N, int Np, int BATCH)
// {
//     int jump{(N + Np) / 2};
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int idz = blockIdx.z * blockDim.z + threadIdx.z;

//     if ((idx < Np) && (idy < N) && (idz < BATCH))
//     {
//         output[idz * Np * N + idy * Np + idx] = input[idz * jump + idx + idy];
//     }
// }

// void __global__ fast_create_center_batch_gpu(hipfftComplex *input, hipfftComplex *output, int N, int Np, int BATCH)
// {
//     int jump{(N + Np) / 2};
//     int jump_mid{Np / 2};
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;

//     if ((idx < N) && (idy < BATCH))
//     {
//         output[idy * N + idx] = input[idy * jump + jump_mid + idx];
//     }
// }

// void __global__ create_center_matrix_gpu(hipfftComplex *input, hipfftComplex *output, int N, int Np)
// {
//     // int jump{(N + Np) / 2};
//     int jump_mid{Np / 2};

//     int idx = blockIdx.x * blockDim.x + threadIdx.x;

//     if ((idx < N))
//     {
//         output[idx] = input[jump_mid + idx];
//     }
// }

// void __global__ mat_multiply_batch(hipfftComplex *left, hipfftComplex *right, int N, int Np, int BATCH)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int idz = blockIdx.z * blockDim.z + threadIdx.z;
//     int id = idz * N * Np + idy * Np + idx;
//     if ((idx < Np) && (idy < N) && (idz < BATCH))
//     {
//         right[id] = make_hipComplex(left[id].x * right[id].x - left[id].y * right[id].y, left[id].x * right[id].y + left[id].y * right[id].x);
//     }
// }

// void __global__ fft_shift_batch(bool stride, hipfftComplex *inp, int N, int Np, int BATCH)
// {
//     int sign;
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int idz = blockIdx.z * blockDim.z + threadIdx.z;
//     int id = idy * Np + idx;
//     int id3d = idz * N * Np + id;
//     if ((idx < Np) && (idy < N) && (idz < BATCH))
//     {
//         if (stride)
//         {
//             sign = 1 - 2 * (id % 2);
//         }
//         else
//         {
//             int row_pos = (int)id / Np;
//             sign = 1 - 2 * (row_pos % 2);
//         }
//         inp[id3d] = make_hipComplex(inp[id3d].x * static_cast<float>(sign), inp[id3d].y * static_cast<float>(sign));
//     }
// }

// void __global__ mat_mul_fft_shift_batch(bool stride, hipfftComplex *left, hipfftComplex *right, int N, int Np, int BATCH)
// {
//     int sign;
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int idz = blockIdx.z * blockDim.z + threadIdx.z;
//     int id3d = idz * N * Np + idy * Np + idx;
//     int id = idy * Np + idx;
//     if ((idx < Np) && (idy < N) && (idz < BATCH))
//     {
//         if (stride)
//         {
//             sign = 1 - 2 * (id % 2);
//         }
//         else
//         {
//             int row_pos = (int)id / Np;
//             sign = 1 - 2 * (row_pos % 2);
//         }
//         right[id3d] = make_hipComplex((left[id3d].x * right[id3d].x - left[id3d].y * right[id3d].y) * static_cast<float>(sign), (left[id3d].x * right[id3d].y + left[id3d].y * right[id3d].x) * static_cast<float>(sign));
//     }
// }

// Called with stride = true
void __global__ mat_mul_fft_shift_batch_reshape(hipfftComplex *window, hipfftComplex *input, hipfftComplex *out, int N, int Np, int BATCH)
{
    int jump{(N + Np) / 2};
    int sign;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    int id3d = idz * N * Np + idy * Np + idx;
    int id = idy * Np + idx;

    int input_id = idz * jump + idx + idy;
    int window_id = idx;

    if ((idx < Np) && (idy < N) && (idz < BATCH))
    {
        sign = 1 - 2 * (id % 2);
        // else
        // {
        //     int row_pos = (int)id / Np;
        //     sign = 1 - 2 * (row_pos % 2);
        // }
        out[id3d] = make_hipComplex((input[input_id].x * window[window_id].x - input[input_id].y * window[window_id].y) * static_cast<float>(sign), (input[input_id].x * window[window_id].y + input[input_id].y * window[window_id].x) * static_cast<float>(sign));
    }
}

// // This reshape function has been tested to be correct
// void __global__ reshape(hipfftComplex *inp, hipfftComplex *output, int nrow, int ncoloumn)
// {
//     __shared__ hipfftComplex tile[BLOCKCHANNELS][BLOCKSLICES];
//     int input_x_coord = blockIdx.x * blockDim.x + threadIdx.x;
//     int input_y_coord = blockIdx.y * blockDim.y + threadIdx.y;
//     // int z_coord = blockIdx.z * blockDim.z + threadIdx.z;
//     auto inter = inp + ncoloumn * input_y_coord + input_x_coord;
//     tile[threadIdx.x][threadIdx.y] = *inter;
//     __syncthreads();
//     int output_grid_y_coord = (blockIdx.x * blockDim.x + threadIdx.y) * nrow;
//     int output_grid_x_coord = blockIdx.y * blockDim.y + threadIdx.x;
//     auto outer = output + output_grid_y_coord + output_grid_x_coord;
//     (*outer) = tile[threadIdx.y][threadIdx.x];
// }

// void __global__ reshape(hipfftComplex *inp, hipfftComplex *output, int nrow, int ncoloumn, int batch)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int idz = blockIdx.z * blockDim.z + threadIdx.z;

//     int input_id = idz * nrow * ncoloumn + idy * 
// }

// void __global__ mat_vec_multiply_fft_shift_batch(bool stride, hipfftComplex *left_mat, hipfftComplex *left_vec, hipfftComplex *right, bool conj, int N, int Np, int BATCH)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int idz = blockIdx.z * blockDim.z + threadIdx.z;

//     int id1d = idz * N + idy;
//     int id2d = idy * Np + idx;
//     int id3d = idz * N * Np + id2d;
//     int sign;
//     // int id = idz * N * Np + idy * Np + idx;
//     hipfftComplex intermediate;
//     if ((idx < Np) && (idy < N) && (idz < BATCH))
//     {
//         if (stride)
//         {
//             sign = 1 - 2 * (id2d % 2);
//         }
//         else
//         {
//             int row_pos = (int)id2d / Np;
//             sign = 1 - 2 * (row_pos % 2);
//         }
//         if (conj)
//             intermediate = make_hipComplex(left_mat[id3d].x * left_vec[id1d].x + left_mat[id3d].y * left_vec[id1d].y, -left_mat[id3d].x * left_vec[id1d].y + left_mat[id3d].y * left_vec[id1d].x);
//         else
//         {
//             intermediate = make_hipComplex(left_mat[id3d].x * left_vec[id1d].x - left_mat[id3d].y * left_vec[id1d].y, left_mat[id3d].x * left_vec[id1d].y + left_mat[id3d].y * left_vec[id1d].x);
//         }
//         right[id3d] = make_hipComplex((intermediate.x * right[id3d].x - intermediate.y * right[id3d].y) * static_cast<float>(sign), (intermediate.x * right[id3d].y + intermediate.y * right[id3d].x) * static_cast<float>(sign));
//     }
// }

// Called with stride = false
// left_mat should have dimensions N*Np and contains the product of exp_mat and kaiser_2
void __global__ mat_vec_multiply_fft_shift_batch_center(hipfftComplex *left_mat, hipfftComplex* input, hipfftComplex *right, bool conj, int N, int Np, int BATCH)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // int id1d = idz * N + idy;
    int id2d = idy * Np + idx;
    int id3d = idz * N * Np + id2d;
    int sign;
    // int id = idz * N * Np + idy * Np + idx;

    int jump{(N + Np) / 2};
    int jump_mid{Np / 2};

    int id_jump = idz * jump + jump_mid + idy;

    hipfftComplex intermediate;
    if ((idx < Np) && (idy < N) && (idz < BATCH))
    {
        // if (stride)
        // {
        //     sign = 1 - 2 * (id2d % 2);
        // }
        // else
        // {
        int row_pos = (int)id2d / Np;
        sign = 1 - 2 * (row_pos % 2);
        // }
        if (conj)
            intermediate = make_hipComplex(left_mat[id2d].x * input[id_jump].x + left_mat[id2d].y * input[id_jump].y, -left_mat[id2d].x * input[id_jump].y + left_mat[id2d].y * input[id_jump].x);
        else
        {
            intermediate = make_hipComplex(left_mat[id2d].x * input[id_jump].x - left_mat[id2d].y * input[id_jump].y, left_mat[id2d].x * input[id_jump].y + left_mat[id2d].y * input[id_jump].x);
        }
        right[id3d] = make_hipComplex((intermediate.x * right[id3d].x - intermediate.y * right[id3d].y) * static_cast<float>(sign), (intermediate.x * right[id3d].y + intermediate.y * right[id3d].x) * static_cast<float>(sign));
    }
}

// void __global__ vect_multiply_batch(hipfftComplex *in, hipfftComplex *vec, bool conj, int N, int Np, int BATCH)
// {
//     // int id = blockIdx.x * blockDim.x + threadIdx.x;
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int idz = blockIdx.z * blockDim.z + threadIdx.z;

//     int id1d = idz * N + idy;
//     int id2d = idy * Np + idx;
//     int id3d = idz * N * Np + id2d;
//     // int vec_ind;
//     if ((idx < Np) && (idy < N) && (idz < BATCH))
//     {
//         if (conj)
//             in[id3d] = make_hipComplex(in[id3d].x * vec[id1d].x + in[id3d].y * vec[id1d].y, -in[id3d].x * vec[id1d].y + in[id3d].y * vec[id1d].x);
//         else
//         {
//             in[id3d] = make_hipComplex(in[id3d].x * vec[id1d].x - in[id3d].y * vec[id1d].y, in[id3d].x * vec[id1d].y + in[id3d].y * vec[id1d].x);
//         }
//     }
// }

// void create_batched_center(hipfftComplex *input, hipfftComplex *output, int N, int Np, int BATCH)
// {
//     int jump_mid{Np / 2};
//     int jump_start{(N + Np) / 2}; // bool conj;
//     // complex<float> *load_buffer;
//     for (int id = 0; id < BATCH; id++)
//         hipMemcpy(output + id * N, input + (id * jump_start + jump_mid), sizeof(hipfftComplex) * N, hipMemcpyDeviceToDevice);
// }

ssca_cuda::ssca_cuda(complex<float> *k1, complex<float> *e_mat, int Nval, int Npval, int size_val)
{
    N = Nval;
    Np = Npval;
    BATCH = 2 * size_val / (Nval + Npval);
    size = size_val;
    rank = 1;
    istride_1 = 1;
    idist_1 = Np;
    batch_1 = N * BATCH;
    ostride_1 = 1;
    odist_1 = Np;
    istride_2 = Np;
    idist_2 = 1;
    batch_2 = Np;
    odist_2 = 1;
    ostride_2 = Np;
    // istride_2 = 1;
    // idist_2 = N;
    // batch_2 = Np*BATCH;
    // odist_2 = N;
    // ostride_2 = 1;
    n_1 = new int[1];
    n_2 = new int[1];
    *n_1 = Np;
    *n_2 = N;
    inembed_1 = n_1;
    onembed_1 = n_1;
    inembed_2 = n_2;
    onembed_2 = n_2;

    streams = new hipStream_t[BATCH];
    // // plans_1 = new hipfftHandle [BATCH];
    plans_2 = new hipfftHandle[BATCH];
    events = new hipEvent_t[BATCH];

    hipfftPlanMany(&plan_1, rank, n_1, inembed_1, istride_1, idist_1, onembed_1, ostride_1, odist_1, HIPFFT_C2C, batch_1);
    // hipfftPlanMany(&plan_2, rank, n_2, inembed_2, istride_2, idist_2, onembed_2, ostride_2, odist_2, HIPFFT_C2C, batch_2);

    for (int i = 0; i < BATCH; i++)
    {
        hipStreamCreate(&streams[i]);
        hipEventCreate(&events[i]);
        hipfftPlanMany(&plans_2[i], rank, n_2, inembed_2, istride_2, idist_2, onembed_2, ostride_2, odist_2, HIPFFT_C2C, batch_2);
        // hipfftPlanMany(&plans_2[i], rank, n_2, inembed_2, istride_2, idist_2, onembed_2, ostride_2, odist_2, HIPFFT_C2C, batch_2);
        // hipfftSetStream(plans_1[i], streams[i]);
        // hipfftSetStream(plans_2[i], streams[i]);
    }
    // hipfftPlanMany(&plan_2, rank, n_2, inembed_2, istride_2, idist_2, onembed_2, ostride_2, odist_2, HIPFFT_C2C, batch_2);
    hipMalloc((void **)&kaiser_1, sizeof(hipfftComplex) * Np * N * BATCH);
    // hipMalloc((void **)&kaiser_2, sizeof(hipfftComplex) * N * Np * BATCH);
    hipMalloc((void **)&exp_mat, sizeof(hipfftComplex) * N * Np * BATCH);
    hipMemcpy(exp_mat, e_mat, sizeof(hipfftComplex) * N * Np, hipMemcpyHostToDevice);
    // hipMemcpy(kaiser_1, k1, sizeof(hipfftComplex) * Np * N * BATCH, hipMemcpyHostToDevice);
    hipMemcpy(kaiser_1, k1, sizeof(hipfftComplex) * Np, hipMemcpyHostToDevice);
    // hipMemcpy(kaiser_2, k2, sizeof(hipfftComplex) * N * N * BATCH, hipMemcpyHostToDevice);
    hipMalloc((void **)&inp_buffer, sizeof(hipfftComplex) * size);
    hipMalloc((void **)&inter_gpu, sizeof(hipfftComplex) * N * Np * BATCH);
    // hipMalloc((void **)&inter_inter_gpu, sizeof(hipfftComplex) * N * Np * BATCH);
    // hipMalloc((void **)&inter_center_gpu, sizeof(hipfftComplex) * N * BATCH);
    hipMalloc((void **)&output_buffer, sizeof(float) * N * Np);
}

void ssca_cuda::cyclo_gram(hipfftComplex *input, float *output, bool conj)
{
    dim3 dimBlockMatrixCreation(16, 16, 4);
    dim3 dimGridMatrixCreation(Np / 16, N / 16, BATCH / 4);

    // dim3 dimBlockCenterCreation(32, 32);
    // dim3 dimGridCenterCreation(N / 32, BATCH / 32);

    // dim3 dimBlockReshape(32, 32);
    // dim3 dimGridReshape(Np / 32, N / 32);

    // dim3 dimBlockReshapeAgain(32, 32);
    // dim3 dimGridReshapeAgain(N / 32, Np / 32);

    hipMemcpy(inp_buffer, input, sizeof(hipfftComplex) * size, hipMemcpyHostToDevice);
    mat_mul_fft_shift_batch_reshape<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(kaiser_1, inp_buffer, inter_gpu, N, Np, BATCH);
    // create_batch_matrix_gpu_3d<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(inp_buffer, inter_gpu, N, Np, BATCH);
    // fast_create_center_batch_gpu<<<dimGridCenterCreation, dimBlockCenterCreation>>>(inp_buffer, inter_center_gpu, N, Np, BATCH);
    // mat_multiply_batch<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(kaiser_1, inter_gpu, N, Np, BATCH);
    // fft_shift_batch<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(true, inter_gpu, N, Np, BATCH);
    // mat_mul_fft_shift_batch<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(true, kaiser_1, inter_gpu, N, Np, BATCH);
    auto arr = hipfftExecC2C(plan_1, inter_gpu, inter_gpu, HIPFFT_FORWARD);
    // mat_multiply_batch<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(exp_mat, inter_gpu, N, Np, BATCH);
    // // mat_multiply_batch<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(kaiser_2, inter_gpu, N, Np, BATCH);
    // // TODO : Combine Kaiser 2 and ExpMat into ExpMat.
    // vect_multiply_batch<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(inter_gpu, inter_center_gpu, conj, N, Np, BATCH);
    // mat_vec_multiply_fft_shift_batch<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(false, exp_mat, inter_center_gpu, inter_gpu, conj, N, Np, BATCH);
    mat_vec_multiply_fft_shift_batch_center<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(exp_mat, inp_buffer, inter_gpu, conj, N, Np, BATCH);
    // fft_shift_batch<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(false, inter_gpu, N, Np, BATCH);
    // reshape<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(inter_gpu, inter_inter_gpu, N, Np);
    // auto arr_2 = hipfftExecC2C(plan_2, inter_inter_gpu, inter_inter_gpu, HIPFFT_FORWARD);
    // reshape<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(inter_inter_gpu, inter_gpu, Np, N);
    for (int i = 0; i < BATCH; i++)
    {
        // reshape<<<dimGridReshape, dimBlockReshape,0,streams[i]>>>(inter_gpu + i*N*Np, inter_inter_gpu + i*N*Np, N, Np);
        // reshape<<<dimGridReshapeAgain, dimBlockReshapeAgain,0,streams[i]>>>(inter_inter_gpu + i*N*Np, inter_gpu + i*N*Np, N, Np);
        auto err = hipfftExecC2C(plans_2[i], inter_gpu + i*N*Np, inter_gpu + i*N*Np, HIPFFT_FORWARD);
        auto err_3 = hipGetLastError();
        hipEventRecord(events[i], streams[i]);
    }

    for (int i = 0; i < BATCH; i++)
    {
        hipStreamWaitEvent(0, events[i], 0);
    }
    // auto err = hipfftExecC2C(plan_2, inter_inter_gpu, inter_inter_gpu, HIPFFT_FORWARD);
    batch_average<<<dimGridMatrixCreation, dimBlockMatrixCreation>>>(inter_gpu, output_buffer, N, Np, BATCH);
    hipMemcpy(output, output_buffer, sizeof(float) * N * Np, hipMemcpyDeviceToDevice);
}
ssca_cuda::~ssca_cuda()
{
    for (int i = 0; i < BATCH; i++)
    {
        hipfftDestroy(plans_2[i]);
        // hipfftDestroy(plans_1[i]);
        hipEventDestroy(events[i]);
        hipStreamDestroy(streams[i]);
    }
    hipfftDestroy(plan_1);
    // hipfftDestroy(plan_2);
    delete[] n_1;
    delete[] n_2;

    // delete[] streams;
    // delete[] events;
    // // delete[] plans_1;
    // delete[] plans_2;
    hipFree(exp_mat);
    hipFree(kaiser_1);
    // hipFree(kaiser_2);
    // hipFree(exp_mat_gpu);
    hipFree(inter_gpu);
    // hipFree(inter_inter_gpu);
    hipFree(inp_buffer);
    hipFree(output_buffer);
    // hipFree(inter_center_gpu);
    // hipFree(kaiser_1_gpu);
    // hipFree(kaiser_2_gpu);
}
